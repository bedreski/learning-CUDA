#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel() {
    printf("Hello World!\n");
}


int main() {
    dkernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}

//Compile: nvcc hello.cu
//Run: ./a.out
